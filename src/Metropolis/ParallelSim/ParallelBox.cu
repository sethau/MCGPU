/*
	Implements methods related to managing data between the host and device.
	Subclass of Box.
*/

#include "ParallelBox.cuh"
#include "ParallelCalcs.cuh"
#include "Metropolis/SerialSim/SerialCalcs.h"

using namespace std;

ParallelBox::ParallelBox(): Box()
{
	nextMol = 0;
	numChanged = 0;
}

ParallelBox::~ParallelBox()
{
	// TODO: free device memory
}

int ParallelBox::chooseMolecules(const int N)
{
	if (changedIndices == NULL || changedMols == NULL)
	{
		changedIndices = new int[N];
		changedMols = new Molecule[N];
	}
	
	if (N > 0)
	{
		int i = 0, j;
		
		//if the size needs to change
		if (numChanged != N)
		{
			delete[] changedIndices;
			delete[] changedMols;
			changedIndices = new int[N];
			changedMols = new Molecule[N];
			numChanged = N;
		}
		
		//if the next molecule has already been chosen
		if (nextMol != 0)
		{
			changedIndices[i++] = nextMol;
			nextMol = 0;
		}
		
		for (i = i; i < N; i++)
		{
			nextMol = chooseMolecule();
			
			//make sure this molecule is not already in the batch
			for (j = 0; j < i; j++)
			{
				//if this molecule is already in the batch,
				//stop here and start with this molecule next time
				if (changedIndices[j] == nextMol)
				{
					numChanged = i;
					return numChanged;
				}
			}
			changedIndices[i] = nextMol;
		}
		//if batch had no duplicates, reset nextMol to 0
		nextMol = 0;
		return N;
	}
	else
	{
		return 0;
	}
}

void ParallelBox::changeMolecules()
{
	for (nextChangeIdx = 0; nextChangeIdx < numChanged; nextChangeIdx++)
	{
		changeMolecule(changedIndices[nextChangeIdx]);
	}
}

void ParallelBox::saveChangedMol(int molIdx)
{
	Molecule *sourceMol = &molecules[molIdx];

	//free memory of changedMol before allocate memory
	deleteMolMemberArrays(changedMols + nextChangeIdx);

	memcpy(changedMols + nextChangeIdx, sourceMol, sizeof(Molecule));

	createMolMemberArrays(changedMols + nextChangeIdx, sourceMol);

	copyMolecule(changedMols + nextChangeIdx, sourceMol);
}

void ParallelBox::createMolMemberArrays(Molecule *mol, Molecule *sourceMol)
{
	mol->atoms = new Atom[sourceMol->numOfAtoms];
	mol->bonds = new Bond[sourceMol->numOfBonds];
	mol->angles = new Angle[sourceMol->numOfAngles];
	mol->dihedrals = new Dihedral[sourceMol->numOfDihedrals];
	mol->hops = new Hop[sourceMol->numOfHops];
}

void ParallelBox::deleteMolMemberArrays(Molecule *mol)
{
	delete[] mol->atoms;
	delete[] mol->bonds;
	delete[] mol->angles;
	delete[] mol->dihedrals;
	delete[] mol->hops;
}

void ParallelBox::toggleChange(int changeIdx)
{
	//create temporary copy of backup Molecule
	Molecule *backupMol = (Molecule*) malloc(sizeof(Molecule));
	createMolMemberArrays(backupMol, changedMols + changeIdx);
	copyMolecule(backupMol, changedMols + changeIdx);
	
	//save working copy
	nextChangeIdx = changeIdx;
	saveChangedMol(changedIndices[changeIdx]);
	nextChangeIdx = 0;
	
	//overwrite working copy with backup
	copyMolecule(molecules + changedIndices[changeIdx], backupMol);
	
	//de-allocate temporary copy of backup
	deleteMolMemberArrays(backupMol);
}

int ParallelBox::changeMolecule(int molIdx)
{
	Box::changeMolecule(molIdx);
	writeChangeToDevice(molIdx);
	
	return molIdx;
}

int ParallelBox::rollback(int molIdx)
{
	Box::rollback(molIdx);
	writeChangeToDevice(molIdx);
	
	return molIdx;
}

void ParallelBox::copyDataToDevice()
{
	//create AtomData on host, and fill atomic data arrays on device
	atomsH = new AtomData(atoms, atomCount);
	hipMalloc(&xD, atomCount * sizeof(Real));
	hipMalloc(&yD, atomCount * sizeof(Real));
	hipMalloc(&zD, atomCount * sizeof(Real));
	hipMalloc(&sigmaD, atomCount * sizeof(Real));
	hipMalloc(&epsilonD, atomCount * sizeof(Real));
	hipMalloc(&chargeD, atomCount * sizeof(Real));
	hipMemcpy(xD, atomsH->x, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(yD, atomsH->y, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(zD, atomsH->z, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(sigmaD, atomsH->sigma, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(epsilonD, atomsH->epsilon, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(chargeD, atomsH->charge, atomCount * sizeof(Real), hipMemcpyHostToDevice);
	
	//create device AtomData struct with pointers to filled-in atomic data arrays
	AtomData *tempAD = (AtomData*) malloc(sizeof(AtomData));
	tempAD->x = xD;
	tempAD->y = yD;
	tempAD->z = zD;
	tempAD->sigma = sigmaD;
	tempAD->epsilon = epsilonD;
	tempAD->charge = chargeD;
	tempAD->atomCount = atomsH->atomCount;
	hipMalloc(&atomsD, sizeof(AtomData));
	hipMemcpy(atomsD, tempAD, sizeof(AtomData), hipMemcpyHostToDevice);
	
	//create MoleculeData on host, and fill molecular data arrays on device
	moleculesH = new MoleculeData(molecules, moleculeCount);
	hipMalloc(&atomsIdxD, moleculeCount * sizeof(int));
	hipMalloc(&numOfAtomsD, moleculeCount * sizeof(int));
	hipMemcpy(atomsIdxD, moleculesH->atomsIdx, moleculeCount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(numOfAtomsD, moleculesH->numOfAtoms, moleculeCount * sizeof(int), hipMemcpyHostToDevice);
	
	//create device MoleculeData struct with pointers to filled-in molecular data arrays
	MoleculeData *tempMD = (MoleculeData*) malloc(sizeof(MoleculeData));
	tempMD->atomsIdx = atomsIdxD;
	tempMD->numOfAtoms = numOfAtomsD;
	tempMD->moleculeCount = moleculesH->moleculeCount;
	hipMalloc(&moleculesD, sizeof(MoleculeData));
	hipMemcpy(moleculesD, tempMD, sizeof(MoleculeData), hipMemcpyHostToDevice);
	
	/***********************************************************************
		Without Parallel Steps
	************************************************************************
	
	//data structures for neighbor batch in energy calculation
	nbrMolsH = (int*) malloc(moleculeCount * sizeof(int));
	molBatchH = (int*) malloc(moleculeCount * sizeof(int));
	hipMalloc(&(nbrMolsD), moleculeCount * sizeof(int));
	hipMalloc(&(molBatchD), moleculeCount * sizeof(int));
	
	//upper bound on number of atoms in any molecule
	maxMolSize = 0;
	for (int i = 0; i < moleculesH->moleculeCount; i++)
	{
		if (moleculesH->numOfAtoms[i] > maxMolSize)
		{
			maxMolSize = moleculesH->numOfAtoms[i];
		}
	}
	
	//energies array on device has one segment for each molecule
	//where each segment has the maximum number of
	//possible interatomic energies for one pair of molecules
	energyCount = moleculesH->moleculeCount * maxMolSize * maxMolSize;
	hipMalloc(&(energiesD), energyCount * sizeof(Real));
	
	************************************************************************
		Without Parallel Steps
	***********************************************************************
		With Parallel Steps
	***********************************************************************/
	
	//data structures for neighbor batch in energy calculation
	nbrMolsH = (int*) malloc(ParallelCalcs::MAX_PAR_STEPS * moleculeCount * sizeof(int));
	molBatchH = (int*) malloc(ParallelCalcs::MAX_PAR_STEPS * moleculeCount * sizeof(int));
	hipMalloc(&(nbrMolsD), ParallelCalcs::MAX_PAR_STEPS * moleculeCount * sizeof(int));
	hipMalloc(&(molBatchD), ParallelCalcs::MAX_PAR_STEPS * moleculeCount * sizeof(int));
	
	//upper bound on number of atoms in any molecule
	maxMolSize = 0;
	for (int i = 0; i < moleculesH->moleculeCount; i++)
	{
		if (moleculesH->numOfAtoms[i] > maxMolSize)
		{
			maxMolSize = moleculesH->numOfAtoms[i];
		}
	}
	
	//energies array on device has one segment for each molecule
	//where each segment has the maximum number of
	//possible interatomic energies for one pair of molecules
	energyCount = ParallelCalcs::MAX_PAR_STEPS * moleculesH->moleculeCount * maxMolSize * maxMolSize;
	hipMalloc(&(energiesD), energyCount * sizeof(Real));
	
	/***********************************************************************
		With Parallel Steps
	***********************************************************************/
	
	//initialize energies to 0
	hipMemset(energiesD, 0, energyCount * sizeof(Real));
	
	//copy Environment to device
	hipMalloc(&(environmentD), sizeof(Environment));
	hipMemcpy(environmentD, environment, sizeof(Environment), hipMemcpyHostToDevice);
}

void ParallelBox::writeChangeToDevice(int changeIdx)
{
	//update AtomData atomsH (MoleculeData will not change)
	int startIdx = moleculesH->atomsIdx[changeIdx];
	for (int i = 0; i < molecules[changeIdx].numOfAtoms; i++)
	{
		atomsH->x[startIdx + i] = molecules[changeIdx].atoms[i].x;
		atomsH->y[startIdx + i] = molecules[changeIdx].atoms[i].y;
		atomsH->z[startIdx + i] = molecules[changeIdx].atoms[i].z;
		//sigma, epsilon, and charge will not change, so there is no need to update those arrays
	}

	//copy changed atom data to device
	hipMemcpy(xD + startIdx, atomsH->x + startIdx, moleculesH->numOfAtoms[changeIdx] * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(yD + startIdx, atomsH->y + startIdx, moleculesH->numOfAtoms[changeIdx] * sizeof(Real), hipMemcpyHostToDevice);
	hipMemcpy(zD + startIdx, atomsH->z + startIdx, moleculesH->numOfAtoms[changeIdx] * sizeof(Real), hipMemcpyHostToDevice);
	//sigma, epsilon, and charge will not change, so there is no need to update those arrays
}

bool ParallelBox::changedMolsWithinCutoff(int mol1, int mol2)
{
	Atom atom1 = molecules[changedIndices[mol1]].atoms[environment->primaryAtomIndex];
	Atom atom2 = molecules[changedIndices[mol2]].atoms[environment->primaryAtomIndex];
		
	//calculate periodic difference in coordinates
	Real deltaX = SerialCalcs::makePeriodic(atom1.x - atom2.x, environment->x);
	Real deltaY = SerialCalcs::makePeriodic(atom1.y - atom2.y, environment->y);
	Real deltaZ = SerialCalcs::makePeriodic(atom1.z - atom2.z, environment->z);
	
	Real r2 = (deltaX * deltaX) +
				(deltaY * deltaY) + 
				(deltaZ * deltaZ);
	
	return r2 < environment->cutoff * environment->cutoff;
}